#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void print_thread_info()
{

	printf("threadIdx.x : %d threadIdx.y : %d threadIdx.z : %d blockDim.x : %d blockDim.y : %d  blockDim.z : %d gridDim.x : %d gridDim.y : %d gridDim.z : %d\n",
		threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y, blockDim.z,
		gridDim.x, gridDim.y, gridDim.z);
}


int main()
{	
	int nx, ny;

	nx = 16;
	ny = 16;

	dim3 block(8, 8);
	dim3 grid(nx / block.x, ny / block.y);

	print_thread_info << < grid,block >> > ();
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}

